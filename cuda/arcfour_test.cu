#include "hip/hip_runtime.h"
/*********************************************************************
* Filename:   arcfour_test.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Performs known-answer tests on the corresponding ARCFOUR
              implementation. These tests do not encompass the full
              range of available test vectors, however, if the tests
              pass it is very, very likely that the code is correct
              and was compiled properly. This code also serves as
              example usage of the functions.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <string.h>
#include <sys/stat.h>
#include <hip/hip_runtime.h>
#include "arcfour.h"

/*********************** FUNCTION DEFINITIONS ***********************/
__global__ void block_arcfour(BYTE *data, BYTE *encrypted_data, BYTE *buf, int TAM_BLOCK, int n)
{
	int i;
    int idx = (blockDim.x * blockIdx.x + threadIdx.x)) * TAM_BLOCK;
    BYTE *input = &data[idx];
    BYTE *output = &encrypted_data[idx];
    int leng = TAM_BLOCK;
    if ((idx + TAM_BLOCK) > n)
    	leng = n - idx;

   for (i = 0; i < leng; i++)
   	output[i] = input[i] ^ buf[i];
}

int rc4_device_test_file(char* filename, int nthreads, char* key)
{
    BYTE *data, *buf, *encrypted_data, *decrypted_data;
    BYTE *d_data, *d_buf, *d_encrypted_data, *d_decrypted_data;
    BYTE state[256];
    int pass = 1;
    int n = strlen(filename);
    int i;
    int TAM_BLOCK = 1024;

    int threadsPerBlock = nthreads;
    int blocksPerGrid;

    struct stat st;
    hipError_t err;

    if (stat(filename, &st) == 0){
      data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
      err = hipMalloc((void**) &d_data, sizeof(BYTE) * st.st_size);
    };

    buf = (BYTE *) malloc(sizeof(BYTE) * TAM_BLOCK);
    err = hipMalloc((void**) &d_buf, sizeof(BYTE) * TAM_BLOCK);
    FILE *file = fopen(filename, "rb");
    
    if (data != NULL && file) {
        int current_byte = 0;
        char filename_enc[80], filename_dec[80];
      
        strncpy(filename_enc, filename, n-4);
        filename_enc[n-4] = '\0';
        strcpy(filename_dec, filename_enc);        
        strcat(filename_enc, "_enc");
        strcat(filename_dec, "_dec");

        // grava a extensao em ext
        for (i = 0; i < 5; i++) {
            filename_enc[n + i] = filename[n-4+i];
            filename_dec[n + i] = filename[n-4+i];
        }
 
        FILE *enc_file = fopen(filename_enc, "wb+");
        FILE *dec_file = fopen(filename_dec, "wb+");

        while(fread(&data[current_byte], sizeof(BYTE), 1, file) == 1){
          current_byte += 1;
        };
        n = current_byte;
        blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

        err = hipMemcpy(d_data, data, n, hipMemcpyHostToDevice);

        encrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);
        decrypted_data = (BYTE *) malloc(sizeof(BYTE) * st.st_size);

        err = hipMalloc((void**) &d_encrypted_data, sizeof(BYTE) * st.st_size);
        err = hipMalloc((void**) &d_decrypted_data, sizeof(BYTE) * st.st_size);

        arcfour_key_setup(state, (BYTE *)key, strlen(key));    
        arcfour_generate_stream(state, buf, TAM_BLOCK);

        err = hipMemcpy(d_buf, buf, TAM_BLOCK, hipMemcpyHostToDevice);
        
        /*for (i = 0; i < n; i += TAM_BLOCK) {
          int leng = TAM_BLOCK;
          if ((i + TAM_BLOCK) > n)
            leng = n - i;
          block_arcfour(&data[i], &encrypted_data[i], buf, leng);
        }*/

        block_arcfour<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_encrypted_data, d_buf, TAM_BLOCK, n);
        err = hipGetLastError();
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
          printf("Failed (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
        }
        
        err = hipMemcpy(encrypted_data, d_encrypted_data, n, hipMemcpyDeviceToHost);
        fwrite(encrypted_data, sizeof(BYTE), n, enc_file);
        
        block_arcfour<<<blocksPerGrid, threadsPerBlock>>>(d_encrypted_data, d_decrypted_data, d_buf, TAM_BLOCK, n);
        err = hipGetLastError();
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
          printf("Failed (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
        }
        err = hipMemcpy(decrypted_data, d_decrypted_data, n, hipMemcpyDeviceToHost);
        fwrite(decrypted_data, sizeof(BYTE), n, dec_file);

        err = hipFree(d_data);
        err = hipFree(d_buf);
        err = hipFree(d_encrypted_data);
        err = hipFree(d_encrypted_data);

        pass = pass && !memcmp(decrypted_data, data, n);


        fclose(enc_file);
        fclose(dec_file);
    }

    fclose(file);

    return pass;
}



int rc4_test()
{
    BYTE state[256];
    BYTE key[3][10] = {{"Key"}, {"Wiki"}, {"Secret"}};
    BYTE stream[3][10] = {{0xEB,0x9F,0x77,0x81,0xB7,0x34,0xCA,0x72,0xA7,0x19},
                          {0x60,0x44,0xdb,0x6d,0x41,0xb7},
                          {0x04,0xd4,0x6b,0x05,0x3c,0xa8,0x7b,0x59}};
    int stream_len[3] = {10,6,8};
    BYTE buf[1024];
    int idx;
    int pass = 1;

    // Only test the output stream. Note that the state can be reused.
    for (idx = 0; idx < 3; idx++) {
        arcfour_key_setup(state, (BYTE *)key[idx], strlen(key[idx]));
        arcfour_generate_stream(state, buf, stream_len[idx]);
        pass = pass && !memcmp(stream[idx], buf, stream_len[idx]);
    }

    return(pass);
}

void arcfour_device_test_all_files(int nthreads) {
  int i;
  char filenames[8][80] = {
       "../sample_files/hubble_1.tif", 
       "../sample_files/hubble_2.png",
       "../sample_files/hubble_3.tif",
       "../sample_files/king_james_bible.txt",
       "../sample_files/mercury.png",
       "../sample_files/moby_dick.txt",
       "../sample_files/tale_of_two_cities.txt",
       "../sample_files/ulysses.txt"
  };

  for (i = 0; i < 8; i++)
    printf("ARCFOUR DEVICE test file: %s ==> %s\n", filenames[i], rc4_device_test_file(filenames[i], nthreads, "Secret") ? "SUCCEEDED" : "FAILED");
}

/*int main()
{
    printf("ARCFOUR tests: %s\n", rc4_test_file("sample_files/hubble_1.tif", "Secret") ? "SUCCEEDED" : "FAILED");

    return(0);
}*/

int main (int argc, char** argv)
{
    if (argc != 3) {
        printf("Usage: ./arcfour_device #threads/block\n");
        return -1;
    }

    int nthreads = atoi(argv[1]);

    arcfour_device_test_all_files(nthreads);
    

    return 0;
}